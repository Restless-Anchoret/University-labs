#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>
#include <ctime>

using namespace std;

#define MATRIX_DIM 1000
#define EPS 0.000001
#define THREADS_NUM 64

typedef struct {
	int dim;
	double* elements;
} Matrix;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t getDet(const Matrix matrix);


// Поиск ненулевого столбца, начиная с позиции start в строке с номером row
__global__ void findNonZeroColumn(Matrix m, int row, int * nonZeroCol) {
	if (*nonZeroCol != -1) {
		int col = blockIdx.x * blockDim.x + threadIdx.x + row;
		if (col < m.dim) {
			double elem = m.elements[row * m.dim + col];
			if (*nonZeroCol != -1 && fabs(elem) > EPS) {
				*nonZeroCol = col;
			}
		}
	}
}

// Прибавляем к столбцу другой без 0 на диагонали
__global__ void normalizeColumn(Matrix m, int to, int from) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < m.dim) {
		m.elements[m.dim*row + to] += m.elements[m.dim*row + from];
	}
}

// Делим строку на элемент на главной дагонали
__global__ void divideRow(Matrix m, int row) {
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (col < m.dim) {
		m.elements[m.dim*row + col] = m.elements[m.dim*row + col] / m.elements[m.dim*row + row];
	}
}

// Вычитаем строку diag из всех строк матрицы ниже diag, умножая их на элемент в столбце diag
__global__ void updateMatrix(Matrix m, int diag) {
	int subdim = m.dim - diag - 1;
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int row = index / subdim + diag + 1;
	int col = index % subdim + diag + 1;
	if (row < m.dim && col < m.dim) {
		double val = m.elements[m.dim*row + col];
		double diff = m.elements[m.dim*row + diag] * m.elements[m.dim*diag + col];
		m.elements[m.dim*row + col] = val - diff;
	}
}

// Зануляем столбец матрицы ниже заданного элемента
__global__ void putZeros(Matrix m, int diag) {
	int row = blockIdx.x*blockDim.x + threadIdx.x + diag + 1;
	if (row < m.dim) {
		m.elements[m.dim*row + diag] = 0;
	}
}

int main()
{
	srand(time(0));
	unsigned int beginTime = clock();
	Matrix m;
	m.dim = MATRIX_DIM;

	m.elements = new double[MATRIX_DIM*MATRIX_DIM];
	for (int i = 0; i < MATRIX_DIM; i++) {
		for (int j = 0; j < MATRIX_DIM; j++) {
			int index = i*MATRIX_DIM + j;
			if (i == j) {
				m.elements[index] = 1;
			}
			else if (j < i) {
				m.elements[index] = rand();
			}
			else {
				m.elements[index] = 0;
			}
		}
	}
	
	getDet(m);

	cout << "Total time " << clock() - beginTime << endl << endl;

	system("pause");
	return 0;
}

// Получить элемент из матрицы Cuda
double getElemFormcMatrix(Matrix cm, int x, int y) {
	double tmp;
	hipMemcpy(&tmp, cm.elements + x*cm.dim + y, sizeof(double), hipMemcpyDeviceToHost);
	return tmp;
}

// Перевести матрицу из оперативной памяти в матрицу Cuda
Matrix getCudaMatrixByHostMatrix(Matrix matrix) {
	Matrix cMatrix;
	cMatrix.dim = matrix.dim;
	size_t size = matrix.dim * matrix.dim * sizeof(double);
	hipMalloc(&cMatrix.elements, size);
	hipMemcpy(cMatrix.elements, matrix.elements, size, hipMemcpyHostToDevice);
	return cMatrix;
}

// Вычисление определителя
hipError_t getDet(const Matrix matrix)
{
	Matrix cMatrix = getCudaMatrixByHostMatrix(matrix);
	double det = 1;
	hipError_t cudaStatus;

	int *nonZeroColumn;
	hipMalloc(&nonZeroColumn, sizeof(int));


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	double diagElem;

	for (int i = 0; i < MATRIX_DIM; i++) {
		diagElem = getElemFormcMatrix(cMatrix, i, i);

		if (fabs(diagElem) < EPS) {
			int hNonZeroCol = -1;
			int blockNum = ceil((double)(MATRIX_DIM - i) / THREADS_NUM);
			hipMemset(&nonZeroColumn, -1, sizeof(int));
			findNonZeroColumn << <blockNum, THREADS_NUM >> >(cMatrix, i, nonZeroColumn);
			hipDeviceSynchronize();
			hipMemcpy(&hNonZeroCol, nonZeroColumn, sizeof(int), hipMemcpyDeviceToHost);
			diagElem = getElemFormcMatrix(cMatrix, i, hNonZeroCol);
			if (hNonZeroCol != -1 && fabs(diagElem) > EPS) {
				cout << "hNonZeroCol " << hNonZeroCol << endl;
				normalizeColumn << <ceil((double)MATRIX_DIM / THREADS_NUM), THREADS_NUM >> >(cMatrix, i, hNonZeroCol);
				hipDeviceSynchronize();
			} else {
				det = 0;
				cout << "No non zero " << hNonZeroCol << endl;
				break;
			}
		}

		det = det * diagElem;

		if (i != MATRIX_DIM - 1) {
			divideRow << <ceil((double)MATRIX_DIM / THREADS_NUM), THREADS_NUM >> >(cMatrix, i);
			hipDeviceSynchronize();
			
			int columsNum = MATRIX_DIM - i - 1; // число столбцов для обработки
			int rowNum = columsNum; // число строк для обработки
			int threads = columsNum < THREADS_NUM ? columsNum : THREADS_NUM;
			int blocks = ceil(double(rowNum * columsNum) / threads);
			
			updateMatrix << < blocks, threads >> >(cMatrix, i);
			hipDeviceSynchronize();
			
			threads = rowNum < THREADS_NUM ? rowNum : THREADS_NUM;
			blocks = ceil(double(rowNum) / threads);
			
			putZeros << < blocks, threads >> >(cMatrix, i);
			hipDeviceSynchronize();
		}
	}

	cout << "Det = " << det << endl;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d \n", cudaStatus);
	}

	hipFree(&nonZeroColumn);
	hipFree(cMatrix.elements);
	return cudaStatus;
}