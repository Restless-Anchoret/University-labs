#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>
#include <ctime>

using namespace std;

#define MATRIX_DIM 1000
#define EPS 0.000001
#define THREADS_NUM 64

typedef struct {
	int dim;
	double* elements;
} Matrix;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t getDet(const Matrix matrix);


//поиск ненулевого столбца, начиная с позиции start в строке с номером row
__global__ void findNonZeroColumn(Matrix m, int row, int * nonZeroCol) {
	if (*nonZeroCol != -1) {
		int col = blockIdx.x * blockDim.x + threadIdx.x + row;
		if (col < m.dim) {
			double elem = m.elements[row * m.dim + col];
			if (*nonZeroCol != -1 && fabs(elem) > EPS) {
				*nonZeroCol = col;
			}
		}
	}
}

// прибавляем к столбцу другой без 0 на диагонали
__global__ void noralizeColum(Matrix m, int to, int from) {
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	if (row < m.dim) {
		m.elements[m.dim*row + to] += m.elements[m.dim*row + from];
	}
}

// зануляем нижнюю половину столбца
__global__ void updateMatrix(Matrix m, int diag) {
	int subdim = m.dim - diag;
	int row = blockIdx.x*blockDim.x / subdim + diag + 1;
	int col = blockIdx.x*blockDim.x % subdim + threadIdx.x + diag;
	if (col < m.dim) {
		double val = m.elements[m.dim*row + col];
		double diff = m.elements[m.dim*row + diag] * m.elements[m.dim*diag + col];
		__syncthreads();
		m.elements[m.dim*row + col] = val - diff;
	}
}

// делим строку на элемент на главной дагонали
__global__ void divideRow(Matrix m, int row) {
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (col < m.dim) {
		m.elements[m.dim*row + col] = m.elements[m.dim*row + col] / m.elements[m.dim*row + row];
	}
}

int main()
{
	srand(time(0));
	unsigned int beginTime = clock();
	Matrix m;
	m.dim = MATRIX_DIM;
	//m.elements = new double[MATRIX_DIM*MATRIX_DIM] { 0, 2, 0, 1, 4, 3, 1, 5, 10};

	m.elements = new double[MATRIX_DIM*MATRIX_DIM];
	for (int i = 0; i < MATRIX_DIM; i++) {
		for (int j = 0; j < MATRIX_DIM; j++) {
			int index = i*MATRIX_DIM + j;
			if (i == j) {
				m.elements[index] = 1;
			}
			else if (j < i) {
				m.elements[index] = rand();
			}
			else {
				m.elements[index] = 0;
			}
		}
	}
	
	getDet(m);

	cout << "Total time " << clock() - beginTime << endl << endl;

	system("pause");
	return 0;
}

void showMat(double* tmp, string st = "") {
	cout << st << endl;
	for (int i = 0; i < MATRIX_DIM * MATRIX_DIM; i++) {
		cout << tmp[i] << ' ';
	}
	cout << endl;
}

double getElemFormcMatrix(Matrix cm, int x, int y) {
	double tmp;
	hipMemcpy(&tmp, cm.elements + x*cm.dim + y, sizeof(double), hipMemcpyDeviceToHost);
	return tmp;
}

Matrix getCudaMatrixByHostMatrix(Matrix matrix) {
	Matrix cMatrix;
	cMatrix.dim = matrix.dim;
	size_t size = matrix.dim * matrix.dim * sizeof(double);
	hipMalloc(&cMatrix.elements, size);
	hipMemcpy(cMatrix.elements, matrix.elements, size, hipMemcpyHostToDevice);
	return cMatrix;
}

hipError_t getDet(const Matrix matrix)
{
	Matrix cMatrix = getCudaMatrixByHostMatrix(matrix);
	double det = 1;
	hipError_t cudaStatus;
	double tmp[9];

	int *nonZeroColumn;
	hipMalloc(&nonZeroColumn, sizeof(int));


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	double diagElem;

	for (int i = 0; i < MATRIX_DIM; i++) {

		diagElem = getElemFormcMatrix(cMatrix, i, i);

		//hipMemcpy(tmp, cMatrix.elements, sizeof(double) * MATRIX_DIM * MATRIX_DIM, hipMemcpyDeviceToHost);
		//showMat(tmp, "begin");
		if (fabs(diagElem) < EPS) {

			int hNonZeroCol = -1;
			int blockNum = ceil((double)(MATRIX_DIM - i) / THREADS_NUM);
			hipMemset(&nonZeroColumn, -1, sizeof(int));
			findNonZeroColumn << <blockNum, THREADS_NUM >> >(cMatrix, i, nonZeroColumn);
			hipDeviceSynchronize();
			hipMemcpy(&hNonZeroCol, nonZeroColumn, sizeof(int), hipMemcpyDeviceToHost);
			diagElem = getElemFormcMatrix(cMatrix, i, hNonZeroCol);
			if (hNonZeroCol != -1 && fabs(diagElem) > EPS) {
				cout << "hNonZeroCol " << hNonZeroCol << endl;
				noralizeColum << <ceil((double)MATRIX_DIM / THREADS_NUM), THREADS_NUM >> >(cMatrix, i, hNonZeroCol);
				hipDeviceSynchronize();

			}
			else {
				det = 0;
				cout << "No non zero " << hNonZeroCol << endl;
				break;
			}
		}

		det = det * diagElem;
		//cout << "de " <<  diagElem << endl;
		divideRow << <ceil((double)MATRIX_DIM / THREADS_NUM), THREADS_NUM >> >(cMatrix, i);
		hipDeviceSynchronize();

		int columsNum = MATRIX_DIM - i; // число столбцов для обработки
		int rowNum = columsNum - 1; // число строк для обработки
		int threads = columsNum < THREADS_NUM ? columsNum : THREADS_NUM;
		int blocks = rowNum * ceil(double(columsNum) / threads);

		updateMatrix << < blocks, threads >> >(cMatrix, i);
		hipDeviceSynchronize();
	}

	cout << "Det = " << det << endl;

	// Check for any errors launching the kernel
	/*cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	//goto Error;
	}*/

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d \n", cudaStatus);
	}

	hipFree(&nonZeroColumn);
	hipFree(cMatrix.elements);
	return cudaStatus;
}